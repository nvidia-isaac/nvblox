#include "hip/hip_runtime.h"
/*
Copyright 2023 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/integrators/freespace_integrator.h"

#include "nvblox/gpu_hash/internal/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/internal/cuda/gpu_indexing.cuh"
#include "nvblox/integrators/internal/integrators_common.h"
#include "nvblox/utils/timing.h"

namespace nvblox {

static_assert(TsdfBlock::kVoxelsPerSide == FreespaceBlock::kVoxelsPerSide,
              "Need same block dimensions for tsdf and freespace blocks");

// clamp an index in-place to the inclusive range defined by minval and maxval
__device__ void clamp(Index3D& index, int minval, int maxval) {
  for (int i = 0; i < 3; ++i) {
    index(i) = std::max(std::min(index(i), maxval), minval);
  }
}

// Number of padded voxels appended to each side of a block in order to
// allow for filtering. Currently only 1 is supported.
constexpr int kPaddingSize = 1;

// Class for storing a neighborhood of 3x3x3 block pointers. This allows for
// faster lookup of voxels, compared to using a hashmap.
//
// The container is intended to be stored in shared memory and is populated
// collaboratively among the threads in one block
//
// Note that all lookups functions use global block indices, there's thus no
// need to transform the indices at the call site.
template <typename VoxelType>
class BlockNeighborhood {
  using BlockType = VoxelBlock<VoxelType>;

 public:
  // Constants defining the number of block neighbors. These cannot be changed.
  static constexpr int kNumBlocks1D = 3;
  static constexpr int kNumBlocks3D =
      kNumBlocks1D * kNumBlocks1D * kNumBlocks1D;

  // Lookup a block pointer and populate a block in the 3x3x3 neighborhood.
  // Which block to populate is determined by the current threadIdx.
  //
  // @param center_block_index  Index of the center block in the neighborhood
  // @param block_hash          Hash map used to lookup the block pointers
  __device__ void populateBlock(
      const Index3D& center_block_index,
      const Index3DDeviceHashMapType<VoxelBlock<VoxelType>>& block_hash) {
    // This will be set by all threads, but that's alright since they all write
    // the same value.
    topleft_block_index_ = {center_block_index.x() - 1,
                            center_block_index.y() - 1,
                            center_block_index.z() - 1};

    // Let the first few threads populate the block pointers.
    if (threadIdx.x < kNumBlocks1D && threadIdx.y < kNumBlocks1D &&
        threadIdx.z < kNumBlocks1D) {
      const Index3D block_index_to_populate = {
          topleft_block_index_.x() + threadIdx.x,
          topleft_block_index_.y() + threadIdx.y,
          topleft_block_index_.z() + threadIdx.z};

      setBlock(block_index_to_populate,
               getBlockPtr(block_hash, block_index_to_populate));
    }
  }
  // Getters for a block given a global block_index that is part of the
  // neighborhood
  __device__ const BlockType* getBlock(const Index3D& block_index) const {
    return block_ptrs_[getLinearIndex(block_index)];
  }
  __device__ BlockType* getBlock(const Index3D& block_index) {
    return block_ptrs_[getLinearIndex(block_index)];
  }

  // Getters for a voxel given a global block_index that is part of the
  // neighborhood
  __device__ const VoxelType* getVoxel(const Index3D& block_index,
                                       const Index3D& voxel_index) const {
    const BlockType* block_ptr = getBlock(block_index);
    if (block_ptr == nullptr) {
      return nullptr;
    } else {
      return &block_ptr
                  ->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
    }
  }

  __device__ void setVoxel(const Index3D& block_index,
                           const Index3D& voxel_index, const VoxelType& voxel) {
    BlockType* block_ptr = getBlock(block_index);
    if (block_ptr != nullptr) {
      block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()] =
          voxel;
    }
  }

 private:
  // Setter for block pointer
  __device__ void setBlock(const Index3D& block_index, BlockType* block_ptr) {
    block_ptrs_[getLinearIndex(block_index)] = block_ptr;
  }

  // Convert a global block index into an 1D index in block_ptrs_;
  __device__ int getLinearIndex(const Index3D& block_index) const {
    const Index3D local_index = block_index - topleft_block_index_;

    const int linear_index = local_index.x() + kNumBlocks1D * local_index.y() +
                             kNumBlocks1D * kNumBlocks1D * local_index.z();
    assert(linear_index >= 0 && linear_index < kNumBlocks3D);
    return linear_index;
  }

  Index3D topleft_block_index_;
  BlockType* block_ptrs_[kNumBlocks3D];
};

// Stores a block of voxels with one extra voxel of padding on each side, e.g.
// if the block size is 8x8x8, this container has capacity for 10x10x10 voxels.
// The padded voxels will be populated from neighboring blocks. Intended to be
// used to simplify border handling when filtering and is suitable to be stored
// in shared memory.
//
// Note that access functions use non-padded voxel indices, i.e. there is no
// need to compensate for the padding at the call site since this is done
// internally. For example PaddedBlock::at({0,0,0}) will refer to the top-left
// corner in both the padded and non-padded case. Indices that exceeds or
// preceeds the block dimensions will return a voxel in the neighboring block,
// for example {0, -1, 0} will return a voxel from the neighbor in negative-Y
// direction. and {0, 10, 0} will refer to a block in the positive-Y direction.
template <typename VoxelType>
class PaddedBlock {
  using BlockType = VoxelBlock<VoxelType>;

 public:
  // Size definitions. Cannot be changed
  static constexpr int kVoxelsPerSide = TsdfBlock::kVoxelsPerSide;
  static constexpr int kVoxelsPerSidePadded = kVoxelsPerSide + 2 * kPaddingSize;

  // Populate the voxel at voxel_index by looking it up in block_neighbors
  __device__ void populateVoxel(
      const Index3D& voxel_index, const Index3D& center_block_index,
      const BlockNeighborhood<VoxelType>& block_neighbors) {
    // We will write to this padded block index
    const Index3D target_voxel_index =
        Index3D(voxel_index.x() + kPaddingSize, voxel_index.y() + kPaddingSize,
                voxel_index.z() + kPaddingSize);

    // Determine which block/voxel we will read from.
    Index3D source_block_index = center_block_index;
    Index3D source_voxel_index = voxel_index;

    // If we're on the border we need to read the voxel from a neigboring
    // block. Adjust block_index and voxel_index accordingly
    for (int axis = 0; axis < 3; ++axis) {
      // We're at the low boundary. Decrement block index to get previous block
      // along current axis.
      if (source_voxel_index(axis) == -1) {
        --source_block_index(axis);
        source_voxel_index(axis) =
            kVoxelsPerSide - 1;  // Voxel index will wrap around to the end
      } else if (source_voxel_index(axis) == (kVoxelsPerSide)) {
        // We're at the high boundary. Increment block to get next block along
        // current axis
        ++source_block_index(axis);
        source_voxel_index(axis) = 0;  // Voxel index will wrap around to zero
      }
    }

    // Set the voxel if it exists
    const VoxelType* source_voxel_ptr =
        block_neighbors.getVoxel(source_block_index, source_voxel_index);
    if (source_voxel_ptr == nullptr) {
      // if the voxel doesn't exist  (because the block is outside the map) we
      // duplicate an adjacent voxel insted.

      // Sanity check that the input index was really on the border
      assert(!isWithinBlockBounds(voxel_index));
      clamp(source_voxel_index, 0, kVoxelsPerSide - 1);
      source_voxel_ptr =
          block_neighbors.getVoxel(center_block_index, source_voxel_index);
    }

    assert(source_voxel_ptr != nullptr);
    voxels_[target_voxel_index.x()][target_voxel_index.y()]
           [target_voxel_index.z()] = *source_voxel_ptr;
  }

  // Returns true if the voxel lies inside the block i.e. *not* in the padded
  // space
  __device__ bool isWithinBlockBounds(const Index3D& voxel_index) {
    return (voxel_index.x() >= 0 && voxel_index.x() < (kVoxelsPerSide) &&
            voxel_index.y() >= 0 && voxel_index.y() < (kVoxelsPerSide) &&
            voxel_index.z() >= 0 && voxel_index.z() < (kVoxelsPerSide));
  }

  // Access functions.
  __device__ VoxelType& at(const Index3D& voxel_index) {
    return voxels_[voxel_index.x() + kPaddingSize]
                  [voxel_index.y() + kPaddingSize]
                  [voxel_index.z() + kPaddingSize];
  }
  __device__ const VoxelType& at(const Index3D& voxel_index) const {
    return voxels_[voxel_index.x() + kPaddingSize]
                  [voxel_index.y() + kPaddingSize]
                  [voxel_index.z() + kPaddingSize];
  }

 private:
  VoxelType voxels_[kVoxelsPerSidePadded][kVoxelsPerSidePadded]
                   [kVoxelsPerSidePadded];
};  // namespace nvblox

// Return true if the voxel is free according to Dynablox Eq. (10) (single
// voxel)
__device__ bool isVoxelFree(const FreespaceVoxel& freespace_voxel,
                            const TsdfVoxel& tsdf_voxel, Time current_time_ms,
                            Time min_duration_since_occupied_for_freespace_ms) {
  return tsdf_voxel.weight > 1e-6 &&
         freespace_voxel.last_occupied_timestamp_ms <=
             current_time_ms - min_duration_since_occupied_for_freespace_ms;
}

// Return true if all voxels in a neighborhood are free.
__device__ bool isVoxelNeighborhoodFree(
    const Index3D& voxel_index,
    const PaddedBlock<FreespaceVoxel>& freespace_block_padded,
    const PaddedBlock<TsdfVoxel>& tsdf_block_padded, Time current_time_ms,
    Time min_duration_since_occupied_for_freespace_ms) {
  bool neighborhood_is_free = true;

  for (int x = -kPaddingSize; x <= kPaddingSize; x++) {
    for (int y = -kPaddingSize; y <= kPaddingSize; y++) {
      for (int z = -kPaddingSize; z <= kPaddingSize; z++) {
        if (x == 0 && y == 0 && z == 0) {
          continue;  // Do not add the original voxel.
        }

        const Index3D neighbor_index = voxel_index + Index3D(x, y, z);
        const TsdfVoxel& tsdf_voxel = tsdf_block_padded.at(neighbor_index);
        const FreespaceVoxel& freespace_voxel =
            freespace_block_padded.at(neighbor_index);

        neighborhood_is_free &=
            isVoxelFree(freespace_voxel, tsdf_voxel, current_time_ms,
                        min_duration_since_occupied_for_freespace_ms);
      }
    }
  }
  return neighborhood_is_free;
}

__global__ void updateFreespaceLayerKernel(
    const Index3DDeviceHashMapType<TsdfBlock> tsdf_block_hash,
    const Index3D* block_indices_to_update, float voxel_size,
    float max_tsdf_distance_for_occupancy_m,
    Time max_unobserved_to_keep_consecutive_occupancy_ms,
    Time min_duration_since_occupied_for_freespace_ms,
    Time min_consecutive_occupancy_duration_for_reset_ms,
    bool check_neighborhood, Time last_update_time_ms,
    Time current_update_time_ms,
    Index3DDeviceHashMapType<FreespaceBlock> freespace_block_hash) {
  // This kernel implements the freespace update as described in the
  // dynablox paper (https://ieeexplore.ieee.org/document/10218983).
  //
  // It consist of the following steps:
  // - Initialization of freespace voxels if seen for the first time.
  // - Update the consecutive_occupancy_duration_ms field
  // - Update the last_occupied_timestamp_ms field
  // - Check if the voxel (and all its neighbors if check_neighborhood=true)
  //   is/are free
  // - Update the is_high_confidence_freespace field
  // Every ThreadBlock works on one VoxelBlock (blockIdx.y/z should be zero)
  const Index3D block_index = block_indices_to_update[blockIdx.x];

  // Since a thread block also includes padded voxels, we obtain the
  // actual voxel index by subtracting the border size from thread indices.
  const Index3D voxel_index =
      Index3D(threadIdx.z - kPaddingSize, threadIdx.y - kPaddingSize,
              threadIdx.x - kPaddingSize);

  // Lookup all block pointers in a 3x3x3 neighborhood around block_index and
  // store them in shared memory. This saves us from excessive and expensive
  // hashtable lookups.
  __shared__ BlockNeighborhood<FreespaceVoxel> freespace_block_neighbors;
  __shared__ BlockNeighborhood<TsdfVoxel> tsdf_block_neighbors;
  freespace_block_neighbors.populateBlock(block_index, freespace_block_hash);
  tsdf_block_neighbors.populateBlock(block_index, tsdf_block_hash);
  __syncthreads();

  // Populate shared memory with voxels from the current block. We also copy an
  // additional padded voxel layerborder around the block to allow filtering at
  // the border
  __shared__ PaddedBlock<FreespaceVoxel> freespace_block_padded;
  freespace_block_padded.populateVoxel(voxel_index, block_index,
                                       freespace_block_neighbors);

  __shared__ PaddedBlock<TsdfVoxel> tsdf_block_padded;
  tsdf_block_padded.populateVoxel(voxel_index, block_index,
                                  tsdf_block_neighbors);
  __syncthreads();

  // Get the freespace voxel
  FreespaceVoxel* freespace_voxel = &freespace_block_padded.at(voxel_index);

  // Initialization of freespace
  if (freespace_voxel->last_occupied_timestamp_ms == Time(0)) {
    // All voxels are initialized to being occupied
    freespace_voxel->last_occupied_timestamp_ms = current_update_time_ms;
    freespace_voxel->consecutive_occupancy_duration_ms = Time(0);
    freespace_voxel->is_high_confidence_freespace = false;
  } else {
    // Get the corresponding tsdf voxel
    TsdfVoxel* tsdf_voxel = &tsdf_block_padded.at(voxel_index);

    // Update consecutive occupancy duration
    // Note: We use the last_occupied_timestamp_ms from the last update here to
    // start counting the consecutive_occupancy_duration_ms from 0 ms when a
    // voxel was seen occupied. Dynablox Eq. (9)
    if (current_update_time_ms - freespace_voxel->last_occupied_timestamp_ms <=
        max_unobserved_to_keep_consecutive_occupancy_ms) {
      // Voxel was occupied lately
      freespace_voxel->consecutive_occupancy_duration_ms +=
          current_update_time_ms - last_update_time_ms;
    } else {
      // We haven't seen the voxel occupied for some time
      freespace_voxel->consecutive_occupancy_duration_ms = Time(0);
    }

    // Update the last occupied timestamp
    // Dynablox Eq. (8)
    if (tsdf_voxel->distance <= max_tsdf_distance_for_occupancy_m) {
      // We are close to a surface, let's assume the voxel is occupied
      freespace_voxel->last_occupied_timestamp_ms = current_update_time_ms;
    }

    // Check if the voxel is free
    bool is_free =
        isVoxelFree(*freespace_voxel, *tsdf_voxel, current_update_time_ms,
                    min_duration_since_occupied_for_freespace_ms);

    // Synchronize here because the last_occupied_timestamp_ms field of the
    // neighboring voxels could have been updated during this kernel. This is
    // strictly only necessary if check_neighborhood=true, but syncing inside an
    // if-statement is not recommended since it might lead to a deadlock if
    // threads are diverging.
    __syncthreads();

    // Check if neighbors are free as well
    // Dynablox Eq. (10) (neighborhood)
    if (check_neighborhood && is_free &&
        tsdf_block_padded.isWithinBlockBounds(voxel_index)) {
      is_free &= isVoxelNeighborhoodFree(
          voxel_index, freespace_block_padded, tsdf_block_padded,
          current_update_time_ms, min_duration_since_occupied_for_freespace_ms);
    }

    // Update high confidence freespace
    // Dynablox Eq. (12)
    if (freespace_voxel->consecutive_occupancy_duration_ms >=
        min_consecutive_occupancy_duration_for_reset_ms) {
      // There was consecutive occupancy for some time: reset freespace
      freespace_voxel->is_high_confidence_freespace = false;
    } else {
      // Otherwise high confidence freespace is set if the voxel is free
      // and kept if it was high confidence before
      // Dynablox Eq. (11)
      freespace_voxel->is_high_confidence_freespace =
          freespace_voxel->is_high_confidence_freespace || is_free;
    }
  }

  // Copy shared mem back to global
  if (tsdf_block_padded.isWithinBlockBounds(voxel_index)) {
    freespace_block_neighbors.setVoxel(block_index, voxel_index,
                                       *freespace_voxel);
  }
}

FreespaceIntegrator::FreespaceIntegrator()
    : FreespaceIntegrator(std::make_shared<CudaStreamOwning>()) {}

FreespaceIntegrator::FreespaceIntegrator(
    std::shared_ptr<CudaStream> hip_stream)
    : cuda_stream_(hip_stream) {}

float FreespaceIntegrator::max_tsdf_distance_for_occupancy_m() const {
  return max_tsdf_distance_for_occupancy_m_;
}

void FreespaceIntegrator::max_tsdf_distance_for_occupancy_m(float value) {
  max_tsdf_distance_for_occupancy_m_ = value;
}

Time FreespaceIntegrator::max_unobserved_to_keep_consecutive_occupancy_ms()
    const {
  return max_unobserved_to_keep_consecutive_occupancy_ms_;
}

void FreespaceIntegrator::max_unobserved_to_keep_consecutive_occupancy_ms(
    Time value) {
  max_unobserved_to_keep_consecutive_occupancy_ms_ = value;
}

Time FreespaceIntegrator::min_duration_since_occupied_for_freespace_ms() const {
  return min_duration_since_occupied_for_freespace_ms_;
}

void FreespaceIntegrator::min_duration_since_occupied_for_freespace_ms(
    Time value) {
  min_duration_since_occupied_for_freespace_ms_ = value;
}

Time FreespaceIntegrator::min_consecutive_occupancy_duration_for_reset_ms()
    const {
  return min_consecutive_occupancy_duration_for_reset_ms_;
}

void FreespaceIntegrator::min_consecutive_occupancy_duration_for_reset_ms(
    Time value) {
  min_consecutive_occupancy_duration_for_reset_ms_ = value;
}

bool FreespaceIntegrator::check_neighborhood() const {
  return check_neighborhood_;
}

void FreespaceIntegrator::check_neighborhood(bool value) {
  check_neighborhood_ = value;
}

parameters::ParameterTreeNode FreespaceIntegrator::getParameterTree(
    const std::string& name_remap) const {
  const std::string name =
      (name_remap.empty()) ? "freespace_integrator" : name_remap;
  std::function<std::string(const Time&)> time_to_string = [](const Time& t) {
    return std::to_string(static_cast<int64_t>(t));
  };
  using parameters::ParameterTreeNode;
  return ParameterTreeNode(
      name,
      {
          ParameterTreeNode("max_tsdf_distance_for_occupancy_m:",
                            max_tsdf_distance_for_occupancy_m_),
          ParameterTreeNode("max_unobserved_to_keep_consecutive_occupancy_ms:",
                            max_unobserved_to_keep_consecutive_occupancy_ms_,
                            time_to_string),
          ParameterTreeNode("min_duration_since_occupied_for_freespace_ms:",
                            min_duration_since_occupied_for_freespace_ms_,
                            time_to_string),
          ParameterTreeNode("min_consecutive_occupancy_duration_for_reset_ms:",
                            min_consecutive_occupancy_duration_for_reset_ms_,
                            time_to_string),
          ParameterTreeNode("check_neighborhood:", check_neighborhood_),
      });
}

void FreespaceIntegrator::updateFreespaceLayer(
    const std::vector<Index3D>& block_indices_to_update, Time update_time_ms,
    const TsdfLayer& tsdf_layer, FreespaceLayer* freespace_layer_ptr) {
  timing::Timer integration_timer("freespace/integrate");

  // Check inputs
  CHECK_NOTNULL(freespace_layer_ptr);
  CHECK(freespace_layer_ptr->voxel_size() - tsdf_layer.voxel_size() < 1e-4)
      << "Voxel size of tsdf and freespace layer must be equal.";
  if (block_indices_to_update.empty()) {
    return;
  }
  const size_t num_block_to_update = block_indices_to_update.size();
  current_update_time_ms_ = update_time_ms;

  // Allocate missing blocks
  timing::Timer allocate_timer("freespace/integrate/allocate");
  freespace_layer_ptr->allocateBlocksAtIndices(block_indices_to_update,
                                               *cuda_stream_);
  allocate_timer.Stop();

  timing::Timer update_timer("freespace/integrate/update_blocks");

  // Expand the buffers when needed
  if (num_block_to_update > block_indices_to_update_device_.capacity()) {
    constexpr float kBufferExpansionFactor = 1.5f;
    const int new_size =
        static_cast<int>(kBufferExpansionFactor * num_block_to_update);
    block_indices_to_update_device_.reserveAsync(new_size, *cuda_stream_);
  }

  transferBlocksIndicesToDevice(block_indices_to_update, *cuda_stream_,
                                &block_indices_to_update_host_,
                                &block_indices_to_update_device_);

  // Kernel configuration:
  // - One threadBlock per VoxelBlock
  // - NxNxN threads where N is the block side-length in voxels.
  constexpr int kNumThreads1D = TsdfBlock::kVoxelsPerSide + 2 * kPaddingSize;
  const dim3 kThreadsPerBlock(kNumThreads1D, kNumThreads1D, kNumThreads1D);
  const int num_thread_blocks = num_block_to_update;

  updateFreespaceLayerKernel<<<num_thread_blocks, kThreadsPerBlock, 0,
                               *cuda_stream_>>>(
      tsdf_layer.getGpuLayerView().getHash().impl_,           // NOLINT
      block_indices_to_update_device_.data(),                 // NOLINT
      freespace_layer_ptr->voxel_size(),                      // NOLINT
      max_tsdf_distance_for_occupancy_m_,                     // NOLINT
      max_unobserved_to_keep_consecutive_occupancy_ms_,       // NOLINT
      min_duration_since_occupied_for_freespace_ms_,          // NOLINT
      min_consecutive_occupancy_duration_for_reset_ms_,       // NOLINT
      check_neighborhood_,                                    // NOLINT
      last_update_time_ms_,                                   // NOLINT
      current_update_time_ms_,                                // NOLINT
      freespace_layer_ptr->getGpuLayerView().getHash().impl_  // NOLINT
  );
  cuda_stream_->synchronize();
  checkCudaErrors(hipPeekAtLastError());

  last_update_time_ms_ = update_time_ms;
}

}  // namespace nvblox
