#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/rays/sphere_tracer.h"

#include <thrust/device_malloc.h>
#include <thrust/device_new.h>
#include <thrust/device_ptr.h>

#include "nvblox/gpu_hash/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/cuda/gpu_indexing.cuh"
#include "nvblox/utils/timing.h"

namespace nvblox {

/// NOTE(gogojjh): define the template functions
template std::shared_ptr<const DepthImage> SphereTracer::renderImageOnGPU(
    const Camera& camera, const Transform& T_L_C, const TsdfLayer& tsdf_layer,
    const float truncation_distance_m,
    const MemoryType output_image_memory_type,
    const int ray_subsampling_factor);

template std::shared_ptr<const DepthImage> SphereTracer::renderImageOnGPU(
    const CameraPinhole& camera, const Transform& T_L_C,
    const TsdfLayer& tsdf_layer, const float truncation_distance_m,
    const MemoryType output_image_memory_type,
    const int ray_subsampling_factor);

/////////////////////////////////////////////////////
__device__ inline bool isTsdfVoxelValid(const TsdfVoxel& voxel) {
  constexpr float kMinWeight = 1e-4;
  return voxel.weight > kMinWeight;
}

__device__ thrust::pair<float, bool> cast(
    const Ray& ray,                                  // NOLINT
    Index3DDeviceHashMapType<TsdfBlock> block_hash,  // NOLINT
    float truncation_distance_m,                     // NOLINT
    float block_size_m,                              // NOLINT
    int maximum_steps,                               // NOLINT
    float maximum_ray_length_m,                      // NOLINT
    float surface_distance_epsilon_m) {
  // -------------------------------------------------------------------------
  // Approach: Step along the ray until we find the surface, or fail to find a
  // zero crossing.
  // -------------------------------------------------------------------------

  // The sign of the first valid distance we get.
  enum class FirstDistanceType { kPositive, kNegative, kNotYetKnown };
  FirstDistanceType first_valid_distance = FirstDistanceType::kNotYetKnown;

  // t captures the parameter scaling along ray.direction. We assume
  // that the ray is normalized which such that t has units meters.
  float t = 0.0f;
  for (int i = 0; (i < maximum_steps) && (t < maximum_ray_length_m); i++) {
    // Current point to sample
    const Vector3f p_L = ray.origin() + t * ray.direction();

    // Evaluate the distance at this point
    float step;
    TsdfVoxel* voxel_ptr;

    // Try to get a distance from the layer.
    // If we can't get a distance, let's see what to do...
    if (!getVoxelAtPosition(block_hash, p_L, block_size_m, &voxel_ptr) ||
        !isTsdfVoxelValid(*voxel_ptr)) {
      // 1) We weren't in observed space before this, let's step through this
      // (unobserved) shit and hope to hit something allocated.
      if (first_valid_distance == FirstDistanceType::kNotYetKnown) {
        // step forward by the truncation distance
        step = truncation_distance_m;
      }
      // 2) We were in observed space, now we've left it... let's kill this
      // ray, it's risky to continue.
      // Note(alexmillane): The "risk" here is that we've somehow passed
      // through the truncation band. This occurs occasionally. The risk
      // of continuing is that we can then see through an object. It's safer
      // to stop here and hope for better luck in the next frame.
      else {
        return {t, false};
      }
    }
    // We got a valid distance
    else {
      // If this is our first sample in observed space
      if (first_valid_distance == FirstDistanceType::kNotYetKnown) {
        if (voxel_ptr->distance >= 0.0f) {
          first_valid_distance = FirstDistanceType::kPositive;
        } else {
          first_valid_distance = FirstDistanceType::kNegative;
        }
      }

      // If we're looking for Positive->Negative crossing
      if (first_valid_distance == FirstDistanceType::kPositive) {
        // If Distance negative (or close to it)!
        if (voxel_ptr->distance < surface_distance_epsilon_m) {
          // We found a zero crossing. Terminate successfully.
          // We're gonna terminate.
          // First we "refine" the distance by back stepping the (now negative)
          // distance value
          t += voxel_ptr->distance;
          // Output - Success!
          return {t, true};
        }
        // Distance positive! - keep searching
        else {
          // Step by this amount
          step = voxel_ptr->distance;
        }
      }
      // If we're looking for Negative->Positive crossing
      else {  // (first_valid_distance == FirstDistanceType::kNegative)
        // If Distance positive (or close to it)!
        if (voxel_ptr->distance > -surface_distance_epsilon_m) {
          // We "refine" the distance by back stepping the (now positive])
          // distance value
          t -= voxel_ptr->distance;
          // Output - Success!
          return {t, true};
        }
        // Distance Negative
        else {
          // Step by this amount
          step = -voxel_ptr->distance;
        }
      }
    }

    // Step further along the ray
    t += step;
  }
  // Ran out of number of steps or distance... Fail
  return {t, false};
}

__global__ void sphereTracingKernel(
    const Ray ray,                                   // NOLINT
    Index3DDeviceHashMapType<TsdfBlock> block_hash,  // NOLINT
    float* t,                                        // NOLINT
    bool* success_flag,                              // NOLINT
    float truncation_distance_m,                     // NOLINT
    float block_size_m,                              // NOLINT
    int maximum_steps,                               // NOLINT
    float maximum_ray_length_m,                      // NOLINT
    float surface_distance_epsilon_m) {
  const int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx != 0) return;

  thrust::pair<float, bool> res =
      cast(ray, block_hash, truncation_distance_m, block_size_m, maximum_steps,
           maximum_ray_length_m, surface_distance_epsilon_m);

  *t = res.first;
  *success_flag = res.second;
}

__global__ void sphereTracingKernel(
    const Ray* rays_L,                               // NOLINT
    const int num_rays,                              // NOLINT
    Index3DDeviceHashMapType<TsdfBlock> block_hash,  // NOLINT
    Vector3f* points_L,                              // NOLINT
    bool* success_flags,                             // NOLINT
    float truncation_distance_m,                     // NOLINT
    float block_size_m,                              // NOLINT
    int maximum_steps,                               // NOLINT
    float maximum_ray_length_m,                      // NOLINT
    float surface_distance_epsilon_m) {
  // Extract a ray
  // NOTE(alexmillane): We expect this kernel to be called with sufficient
  // threads.
  const int ray_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (ray_idx >= num_rays) {
    return;
  }
  const Ray ray_L = rays_L[ray_idx];

  // Cast
  const thrust::pair<float, bool> result =
      cast(ray_L, block_hash, truncation_distance_m, block_size_m,
           maximum_steps, maximum_ray_length_m, surface_distance_epsilon_m);

  // Reconstruct the 3D point
  if (!result.second) {
    success_flags[ray_idx] = result.second;
    return;
  }
  const Vector3f p_L = ray_L.pointAt(result.first);

  // Write the output
  points_L[ray_idx] = p_L;
  success_flags[ray_idx] = result.second;
}

template <typename CameraType>
__global__ void sphereTracingKernel(
    const CameraType camera,                         // NOLINT
    const Transform T_L_C,                           // NOLINT
    Index3DDeviceHashMapType<TsdfBlock> block_hash,  // NOLINT
    float* image,                                    // NOLINT
    float truncation_distance_m,                     // NOLINT
    float block_size_m,                              // NOLINT
    int maximum_steps,                               // NOLINT
    float maximum_ray_length_m,                      // NOLINT
    float surface_distance_epsilon_m,                // NOLINT
    int ray_subsampling_factor) {
  const int ray_col_idx = threadIdx.x + blockIdx.x * blockDim.x;
  const int ray_row_idx = threadIdx.y + blockIdx.y * blockDim.y;
  // Note: we ensure that this division works cleanly before getting here.
  const int ray_rows = camera.rows() / ray_subsampling_factor;
  const int ray_cols = camera.cols() / ray_subsampling_factor;
  if ((ray_row_idx >= ray_rows) || (ray_col_idx >= ray_cols)) {
    return;
  }

  // Get the image-plane coordinates of where this ray should pass such that it
  // is in the center of the patch it will represent.
  constexpr float kHalf = 1.0f / 2.0f;
  const Index2D ray_indices(ray_col_idx, ray_row_idx);
  const Vector2f pixel_coords =
      (ray_indices * ray_subsampling_factor).cast<float>() +
      kHalf * static_cast<float>(ray_subsampling_factor) * Vector2f::Ones();

  // Get the ray going through this pixel (in layer coordinate)
  const Vector3f ray_direction_C =
      camera.vectorFromImagePlaneCoordinates(pixel_coords).normalized();
  const Ray ray_L(T_L_C.translation(), T_L_C.linear() * ray_direction_C);

  // Cast the ray into the layer
  thrust::pair<float, bool> t_optional =
      cast(ray_L, block_hash, truncation_distance_m, block_size_m,
           maximum_steps, maximum_ray_length_m, surface_distance_epsilon_m);

  // If success, write depth to image, otherwise write -1.
  if (t_optional.second == true) {
    const float depth = t_optional.first * ray_direction_C.z();
    image::access(ray_row_idx, ray_col_idx, ray_cols, image) = depth;
  } else {
    image::access(ray_row_idx, ray_col_idx, ray_cols, image) = -1.0f;
  }
}

SphereTracer::SphereTracer() {
  checkCudaErrors(hipStreamCreate(&tracing_stream_));
}

SphereTracer::~SphereTracer() {
  hipStreamSynchronize(tracing_stream_);
  checkCudaErrors(hipStreamDestroy(tracing_stream_));
}

int SphereTracer::maximum_steps() const { return maximum_steps_; }

float SphereTracer::maximum_ray_length_m() const {
  return maximum_ray_length_m_;
}

float SphereTracer::surface_distance_epsilon_vox() const {
  return surface_distance_epsilon_vox_;
}

void SphereTracer::maximum_steps(int maximum_steps) {
  CHECK_GT(maximum_steps, 0);
  maximum_steps_ = maximum_steps;
}

void SphereTracer::maximum_ray_length_m(float maximum_ray_length_m) {
  CHECK_GT(maximum_ray_length_m, 0);
  maximum_ray_length_m_ = maximum_ray_length_m;
}

void SphereTracer::surface_distance_epsilon_vox(
    float surface_distance_epsilon_vox) {
  CHECK_GT(surface_distance_epsilon_vox, 0);
  surface_distance_epsilon_vox_ = surface_distance_epsilon_vox;
}

bool SphereTracer::castOnGPU(const Ray& ray, const TsdfLayer& tsdf_layer,
                             const float truncation_distance_m,
                             float* t) const {
  constexpr float eps = 1e-5;
  CHECK_NEAR(ray.direction().norm(), 1.0, eps);

  // Get the GPU hash
  GPULayerView<TsdfBlock> gpu_layer_view = tsdf_layer.getGpuLayerView();

  // Allocate space
  float* t_device;
  bool* success_flag_device;
  hipMalloc(&t_device, sizeof(float));
  hipMalloc(&success_flag_device, sizeof(bool));

  // Kernel
  const float surface_distance_epsilon_m =
      surface_distance_epsilon_vox_ * tsdf_layer.voxel_size();
  sphereTracingKernel<<<1, 1, 0, tracing_stream_>>>(
      ray,                             // NOLINT
      gpu_layer_view.getHash().impl_,  // NOLINT
      t_device,                        // NOLINT
      success_flag_device,             // NOLINT
      truncation_distance_m,           // NOLINT
      gpu_layer_view.block_size(),     // NOLINT
      maximum_steps_,                  // NOLINT
      maximum_ray_length_m_,           // NOLINT
      surface_distance_epsilon_m);

  // GPU -> CPU
  hipMemcpyAsync(t, t_device, sizeof(float), hipMemcpyDeviceToHost,
                  tracing_stream_);
  bool success_flag;
  hipMemcpyAsync(&success_flag, success_flag_device, sizeof(bool),
                  hipMemcpyDeviceToHost, tracing_stream_);

  checkCudaErrors(hipStreamSynchronize(tracing_stream_));
  checkCudaErrors(hipPeekAtLastError());

  // Deallocate
  hipFree(t_device);
  hipFree(success_flag_device);

  return success_flag;
}

template <typename CameraType>
std::shared_ptr<const DepthImage> SphereTracer::renderImageOnGPU(
    const CameraType& camera, const Transform& T_L_C,
    const TsdfLayer& tsdf_layer, const float truncation_distance_m,
    const MemoryType output_image_memory_type,
    const int ray_subsampling_factor) {
  CHECK_EQ(camera.width() % ray_subsampling_factor, 0);
  CHECK_EQ(camera.height() % ray_subsampling_factor, 0);
  CHECK(output_image_memory_type != MemoryType::kHost);
  // Output space
  const int image_height = camera.height() / ray_subsampling_factor;
  const int image_width = camera.width() / ray_subsampling_factor;
  // If we get a request for a different size image, reallocate.
  if (!depth_image_ || depth_image_->width() != image_width ||
      depth_image_->height() != image_height ||
      depth_image_->memory_type() != output_image_memory_type) {
    depth_image_ = std::make_shared<DepthImage>(image_height, image_width,
                                                output_image_memory_type);
  }

  // Get the GPU hash
  timing::Timer hash_transfer_timer(
      "color/integrate/sphere_trace/hash_transfer");
  GPULayerView<TsdfBlock> gpu_layer_view = tsdf_layer.getGpuLayerView();
  hash_transfer_timer.Stop();

  // Get metric surface distance epsilon
  const float surface_distance_epsilon_m =
      surface_distance_epsilon_vox_ * tsdf_layer.voxel_size();

  // Kernel
  // Call params
  // - 1 thread per pixel
  // - 8 x 8 threads per thread block
  // - N x M thread blocks get 1 thread per pixel
  constexpr dim3 kThreadsPerThreadBlock(8, 8, 1);
  const dim3 num_blocks(
      depth_image_->cols() / kThreadsPerThreadBlock.y + 1,  // NOLINT
      depth_image_->rows() / kThreadsPerThreadBlock.x + 1,  // NOLINT
      1);
  sphereTracingKernel<<<num_blocks, kThreadsPerThreadBlock, 0,
                        tracing_stream_>>>(
      camera,                          // NOLINT
      T_L_C,                           // NOLINT
      gpu_layer_view.getHash().impl_,  // NOLINT
      depth_image_->dataPtr(),         // NOLINT
      truncation_distance_m,           // NOLINT
      gpu_layer_view.block_size(),     // NOLINT
      maximum_steps_,                  // NOLINT
      maximum_ray_length_m_,           // NOLINT
      surface_distance_epsilon_m,      // NOLINT
      ray_subsampling_factor);
  checkCudaErrors(hipStreamSynchronize(tracing_stream_));
  checkCudaErrors(hipPeekAtLastError());

  return depth_image_;
}

std::pair<device_vector<Vector3f>, device_vector<bool>> SphereTracer::castOnGPU(
    std::vector<Ray>& rays_L, const TsdfLayer& tsdf_layer,
    const float truncation_distance_m) {
  // Inputs
  device_vector<Ray> rays_L_device(rays_L);

  // Output space
  device_vector<Vector3f> intersection_points_L(rays_L.size());
  device_vector<bool> success_flags(rays_L.size());
  success_flags.setZero();

  // Get the GPU hash
  timing::Timer hash_transfer_timer(
      "color/integrate/sphere_trace/hash_transfer");
  GPULayerView<TsdfBlock> gpu_layer_view = tsdf_layer.getGpuLayerView();
  hash_transfer_timer.Stop();

  // Get metric surface distance epsilon
  const float surface_distance_epsilon_m =
      surface_distance_epsilon_vox_ * tsdf_layer.voxel_size();

  // Kernel
  // Call params
  // - 1 thread per pixel
  // - 64 threads per thread block (chosen arbitrarily)
  // - N thread blocks get 1 thread per ray
  constexpr int kThreadsPerThreadBlock = 32;
  const int num_blocks = rays_L.size() / kThreadsPerThreadBlock + 1;
  sphereTracingKernel<<<num_blocks, kThreadsPerThreadBlock, 0,
                        tracing_stream_>>>(
      rays_L_device.data(),            // NOLINT
      rays_L_device.size(),            // NOLINT
      gpu_layer_view.getHash().impl_,  // NOLINT
      intersection_points_L.data(),    // NOLINT
      success_flags.data(),            // NOLINT
      truncation_distance_m,           // NOLINT
      gpu_layer_view.block_size(),     // NOLINT
      maximum_steps_,                  // NOLINT
      maximum_ray_length_m_,           // NOLINT
      surface_distance_epsilon_m);
  checkCudaErrors(hipStreamSynchronize(tracing_stream_));
  checkCudaErrors(hipPeekAtLastError());

  return {std::move(intersection_points_L), std::move(success_flags)};
}

}  // namespace nvblox
