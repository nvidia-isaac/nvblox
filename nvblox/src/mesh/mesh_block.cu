/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/mesh/mesh_block.h"

namespace nvblox {

MeshBlock::MeshBlock(MemoryType memory_type)
    : vertices(memory_type),
      normals(memory_type),
      colors(memory_type),
      triangles(memory_type) {}

void MeshBlock::clear() {
  vertices.clearNoDealloc();
  normals.clearNoDealloc();
  triangles.clearNoDealloc();
  colors.clearNoDealloc();
}

MeshBlock::Ptr MeshBlock::allocate(MemoryType memory_type) {
  return std::make_shared<MeshBlock>(memory_type);
}

MeshBlock::Ptr MeshBlock::allocateAsync(MemoryType memory_type,
                                        const CudaStream&) {
  return allocate(memory_type);
}

size_t MeshBlock::size() const { return vertices.size(); }

size_t MeshBlock::sizeInBytes() const {
  return vertices.size() * sizeof(Vector3f) +  // NOLINT
         normals.size() * sizeof(Vector3f) +   // NOLINT
         colors.size() * sizeof(Color) +       // NOLINT
         triangles.size() * sizeof(int);
}

size_t MeshBlock::capacity() const { return vertices.capacity(); }

void MeshBlock::expandColorsToMatchVertices() {
  colors.reserve(vertices.capacity());
  colors.resize(vertices.size());
}

void MeshBlock::copyFromAsync(const MeshBlock& other,
                              const CudaStream hip_stream) {
  vertices.copyFromAsync(other.vertices, hip_stream);
  normals.copyFromAsync(other.normals, hip_stream);
  colors.copyFromAsync(other.colors, hip_stream);
  triangles.copyFromAsync(other.triangles, hip_stream);
}

void MeshBlock::copyFrom(const MeshBlock& other) {
  copyFromAsync(other, CudaStreamOwning());
}

// Set the pointers to point to the mesh block.
CudaMeshBlock::CudaMeshBlock(MeshBlock* block) {
  CHECK_NOTNULL(block);
  vertices = block->vertices.data();
  normals = block->normals.data();
  triangles = block->triangles.data();
  colors = block->colors.data();

  vertices_size = block->vertices.size();
  triangles_size = block->triangles.size();
}

}  // namespace nvblox
