#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/map/blox.h"
#include "nvblox/map/common_names.h"

namespace nvblox {

// Must be called with:
// - a single block
// - one thread per voxel
__global__ void setColorBlockGray(ColorBlock* block_device_ptr) {
  ColorVoxel* voxel_ptr =
      &block_device_ptr->voxels[threadIdx.z][threadIdx.y][threadIdx.x];
  // NOTE(alexmillane): We don't use alpha channel for blocks (yet). So we just
  // set 0 here.
  voxel_ptr->color.r = 127;
  voxel_ptr->color.g = 127;
  voxel_ptr->color.b = 127;
  voxel_ptr->color.a = 0;
  voxel_ptr->weight = 0.0f;
}

void setColorBlockGrayOnGPUAsync(ColorBlock* block_device_ptr,
                                 const CudaStream& hip_stream) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const dim3 kThreadsPerBlock(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  setColorBlockGray<<<1, kThreadsPerBlock, 0, hip_stream>>>(block_device_ptr);
  checkCudaErrors(hipPeekAtLastError());
}

}  // namespace nvblox
