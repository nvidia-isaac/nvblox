/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/core/internal/error_check.h"

#include <iostream>

namespace nvblox {

void check_cuda(hipError_t result, char const* const func,
                const char* const file, int const line) {
  if (result) {
    std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at "
              << file << ":" << line << " '" << func
              << "'. Error string: " << hipGetErrorString(result) << ".\n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

void check_npp(NppStatus result, char const* const func, const char* const file,
               int const line) {
  if (result) {
    std::cerr << "NPP error = " << result << " at " << file << ":" << line
              << " '" << func << ".\n";
    // Make sure we call CUDA Device Reset before exiting
    hipDeviceReset();
    exit(99);
  }
}

}  // namespace nvblox
